#include "hip/hip_runtime.h"
typedef double float_ty;

// From http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#axzz4meEZrFDA
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double atomicMul(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val *
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
__device__ float atomicMul(float* address, float val)
{
    int* address_as_i = (int*)address;
    int old = *address_as_i, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(val *
                               __int_as_float(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __int_as_float(old);
}
__device__ double atomicAssign(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, val);

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}


__device__ float_ty signum(float_ty x) {
  if (x < 0) {
    return -1;
  } else if (x > 0) {
    return 1;
  } else {
    return 0;
  }
}

__device__ float_ty atanh_(float_ty x) {
  if (x == 1 || x == -1) {
    return signum(x) * 18.714973875118524;
  } else {
    return atanh(x);
  }
}

__device__ int lamIndex(int i, int j, int sz, int rowCount, int colCount, int* offsets) {
  int shift = i*sz;
  int off   = offsets[(j/sz)*colCount + i];
  if (off > -1) {
    return shift + ((off + j) % sz);
  } else {
    return -1;
  }
}

extern "C" __global__ void tanhTransform(float_ty* mLet, float_ty* lam, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int lamIx = lamIndex(i, j, sz, rowCount, colCount, offsets);
  if (lamIx > -1) {
    float_ty v = mLet[(j*colCount) + i];
    mLet[(j*colCount) + i] = tanh(- ((lam[lamIx] - v)/2));
  }
}

extern "C" __global__ void setToOne(float_ty* mLet, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x;
  int j = threadIdx.y;

  int lamIx = lamIndex(i, j, sz, rowCount, colCount, offsets);
  /* if (lamIx > -1) { */
    mLet[(j*colCount) + i] = 1;
  /* } */
}

extern "C" __global__ void makeNonzeroMat(bool* nonzero, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x;
  int j = threadIdx.y;

  nonzero[(j*colCount) + i] = (offsets[(j/sz)*colCount + i] > -1);
}

extern "C" __global__ void insertOnes(float_ty* mLet, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x;
  int j = threadIdx.y;

  if (offsets[((j/sz)*colCount) + i] == -1) {
    mLet[(j*colCount) + i] = 1;
  }
}

// Arraylet matrix coordinates //
extern "C" __global__ void selfProduct(float_ty* mLet, float_ty* newMLet, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.z;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int kStart = threadIdx.x*(colCount/blockDim.x);
  /* int k = blockIdx.x*blockDim.x + threadIdx.x; */

  if (kStart == 0) {
    newMLet[(j*colCount) + i] = 1;
  }
  __syncthreads();

  float_ty prod = 1;
  if (offsets[(j/sz)*colCount + i] > -1) {
    for (int k = kStart; k < (threadIdx.x+1)*(colCount/blockDim.x); ++k) {
      if (k != i && offsets[(j/sz)*colCount + k] > -1) {
        prod *= mLet[(j*colCount) + k];
        /* newMLet[(j*colCount) + i] *= mLet[(j*colCount) + k]; */
      }
    }

    atomicMul(&newMLet[(j*colCount) + i], prod);
  }
}

extern "C" __global__ void selfProductRows(float_ty* mLet, float_ty* newMLet, int rowCount, int colCount, int sz, int* offsets) {
  extern __shared__ float_ty smem[];

  int i = threadIdx.x;
  int j = blockIdx.x;
  /* int k = (threadIdx.x + 1)%blockDim.x; */

  smem[i] = mLet[(j*colCount) + i];
  __syncthreads();

  if (offsets[(j/sz)*colCount + i] > -1) {
    for (int s = 1; s < blockDim.x; s *= 2) {
      if (i % (2*s) == 0 && offsets[(j/sz)*colCount + i + sz] > -1) {
        smem[i] *= smem[i + s];
      }
      __syncthreads();
    }
    /* for (int s = blockDim.x/2; s > 0; s >>= 1) { */
    /*   if (i < s) { */
    /*     /1* smem[i] = smem[(i+1)%blockDim.x]*smem[i+s]; *1/ */
    /*     smem[i] *= smem[i+s]; */
    /*   } */
    /*   __syncthreads(); */
    /* } */
    newMLet[(j*colCount) + i] = smem[0]; // /mLet[(j*colCount) + i];
  }

  /* if (threadIdx.x == 0) { */
  /* atomicMul(&newMLet[(j*colCount) + i], smem[0]/mLet[(j*colCount) + i]); */
  /* } */
}

extern "C" __global__ void atanhTransform(float_ty* newMLet, int rowCount, int colCount, int sz, int* offsets) {
  /* int i = blockIdx.x; */
  /* int j = threadIdx.y; */
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  if (offsets[(j/sz)*colCount + i] > -1) {
    newMLet[(j*colCount) + i] = -2*atanh_(newMLet[(j*colCount)+i]);
  } else {
    newMLet[(j*colCount) + i] = 0;
  }
}

// Arraylet matrix coordinates //
extern "C" __global__ void updateLam(float_ty* newLam, float_ty* newMLet, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int lamIx = lamIndex(i, j, sz, rowCount, colCount, offsets);

  if (lamIx > -1) {
    atomicAdd(&newLam[lamIx], newMLet[(j*colCount) + i]);
  }
}

__device__ bool hard(float_ty v) {
  return v > 0;
}

extern "C" __global__ void parityRowResults(int* rowResults, float_ty* lam, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y;
  if (i == 0) {
    atomicAnd(&rowResults[j], 0);
  }

  int lamIx = lamIndex(i, j, sz, rowCount, colCount, offsets);

  int count = __syncthreads_count(lamIx != -1 && hard(lam[lamIx]));

  if (threadIdx.x == 0) {
    // rowResults[j] = count % 2 == 1;
    atomicAdd(&rowResults[j], count % 2 == 1);
  }
}

// lam vector coordinates //
extern "C" __global__ void checkParity(int* pop, int* rowResults) {
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  if (j == 0) {
    atomicAnd(pop, 0);
  }

  int blockOr = __syncthreads_or(rowResults[j] % 2) != 0 ? 1 : 0;
  atomicOr(pop, blockOr);
}

