#include "hip/hip_runtime.h"
// From http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#axzz4meEZrFDA
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double atomicMul(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val *
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}


__device__ double signum(double x) {
  if (x < 0) {
    return -1;
  } else if (x > 0) {
    return 1;
  } else {
    return 0;
  }
}

__device__ double atanh_(double x) {
  if (x == 1 || x == -1) {
    return signum(x) * 18.714973875118524;
  } else {
    return atanh(x);
  }
}

__device__ int lamIndex(int i, int j, int sz, int rowCount, int colCount, int* offsets) {
  int shift = i*sz;
  int off   = offsets[(j/sz)*colCount + i];
  if (off > -1) {
    return shift + ((off + j) % sz);
  } else {
    return -1;
  }
}

// Arraylet matrix coordinates //
extern "C" __global__ void tanhTransform(double* mLet, double* newMLet, double* lam, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x;
  int j = blockIdx.y;

  int k = threadIdx.x;

  if (k == 0) {
    newMLet[(j*colCount) + i] = 1;
  }
  __syncthreads();

  if (offsets[(j/sz)*colCount + i] > -1) {
    double v = mLet[(j*colCount) + k];

    int lamIx = lamIndex(k, j, sz, rowCount, colCount, offsets);
    if (k != i) {
      if (lamIx > -1) {
        atomicMul(&newMLet[(j*colCount) + i], tanh(- ((lam[lamIx] - v)/2)));
      }
    }
  }
}

extern "C" __global__ void atanhTransform(double* newMLet, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x;
  int j = blockIdx.y;

  if (offsets[(j/sz)*colCount + i] > -1) {
    newMLet[(j*colCount) + i] = -2*atanh_(newMLet[(j*colCount)+i]);
  } else {
    newMLet[(j*colCount) + i] = 0;
  }
}

// Arraylet matrix coordinates //
extern "C" __global__ void updateLam(double* newLam, double* newMLet, int rowCount, int colCount, int sz, int* offsets) {
  int i = blockIdx.x;
  int j = blockIdx.y;
  int lamIx = lamIndex(i, j, sz, rowCount, colCount, offsets);

  if (lamIx > -1) {
    atomicAdd(&newLam[lamIx], newMLet[(j*colCount) + i]);
  }
}

__device__ bool hard(double v) {
  return v > 0;
}

// lam vector coordinates //
extern "C" __global__ void checkParity(int* pop, double* mLet, double* lam, int rowCount, int colCount, int sz, int* offsets) {
  int startJ = threadIdx.y;
  int j      = startJ;

  bool rowResult = false;

  for (int i = 0; i < colCount; ++i) {
    int lamIx = lamIndex(i, j, sz, rowCount, colCount, offsets);

    if (lamIx > -1) {
      rowResult = (rowResult != hard(lam[lamIx]));
    }
  }

  atomicAdd(pop, (rowResult ? 1 : 0));
}

